#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblas.h>

// CUDA kernel for matrix multiplication with A and transpose(B)
__global__ void matMulCudaKernel(float* A, float* B, float* C, int aRows, int aCols, int bRows) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < aRows && col < bRows) {
        float value = 0;
        for (int k = 0; k < aCols; ++k) {
            // B is accessed in transposed manner
            value += A[row * aCols + k] * B[col * aCols + k];
        }
        C[row * bRows + col] = value;
    }
}

extern "C" void matMulCUDA(float* a, int aRows, int aCols, float* b, int bRows, int bCols, float* out) {
    float *d_A, *d_B, *d_C;
    size_t sizeA = aRows * aCols * sizeof(float);
    size_t sizeB = bRows * bCols * sizeof(float);
    size_t sizeC = aRows * bRows * sizeof(float);

    hipMalloc((void**)&d_A, sizeA);
    hipMalloc((void**)&d_B, sizeB);
    hipMalloc((void**)&d_C, sizeC);

    hipMemcpy(d_A, a, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, b, sizeB, hipMemcpyHostToDevice);

    // Cuda Kernel
    dim3 dimBlock(16, 16);
    dim3 dimGrid((bRows + dimBlock.x - 1) / dimBlock.x, (aRows + dimBlock.y - 1) / dimBlock.y);
    matMulCudaKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, aRows, aCols, bRows);

    hipMemcpy(out, d_C, sizeC, hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

// Cublas for matrix multiplication with A and transpose(B)
extern "C" void matMulCublas(float* a, int aRows, int aCols, float* b, int bRows, int bCols, float* out) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float *d_A, *d_B, *d_C;
    size_t sizeA = aRows * aCols * sizeof(float);
    size_t sizeB = bRows * bCols * sizeof(float);
    size_t sizeC = aRows * bRows * sizeof(float);

    hipMalloc((void**)&d_A, sizeA);
    hipMalloc((void**)&d_B, sizeB);
    hipMalloc((void**)&d_C, sizeC);

    hipMemcpy(d_A, a, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, b, sizeB, hipMemcpyHostToDevice);
    
    // Cublas Kernel
    float one = 1.0;
    float zero = 0.0;

    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                bRows, aRows, aCols,
                &one,
                d_B, bCols,
                d_A, aCols,
                &zero, d_C, bRows);

    hipMemcpy(out, d_C, sizeC, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);
}