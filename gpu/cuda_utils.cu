#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblas.h>
#include "cuda_utils.h"

// CUDA kernel for matrix multiplication with A and transpose(B)
__global__ void matMulCudaKernel(float* A, float* B, float* C, int aRows, int aCols, int bRows) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < aRows && col < bRows) {
        float value = 0;
        for (int k = 0; k < aCols; ++k) {
            // B is accessed in transposed manner
            value += A[row * aCols + k] * B[col * aCols + k];
        }
        C[row * bRows + col] = value;
    }
}

extern "C" void matMulCUDA(float* a, int aRows, int aCols, float* b, int bRows, int bCols, float* out) {
    float *d_A, *d_B, *d_C;
    size_t sizeA = aRows * aCols * sizeof(float);
    size_t sizeB = bRows * bCols * sizeof(float);
    size_t sizeC = aRows * bRows * sizeof(float);

    hipMalloc((void**)&d_A, sizeA);
    hipMalloc((void**)&d_B, sizeB);
    hipMalloc((void**)&d_C, sizeC);

    hipMemcpy(d_A, a, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, b, sizeB, hipMemcpyHostToDevice);

    // Cuda Kernel
    dim3 dimBlock(32, 32);
    dim3 dimGrid((bRows + dimBlock.x - 1) / dimBlock.x, (aRows + dimBlock.y - 1) / dimBlock.y);
    matMulCudaKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, aRows, aCols, bRows);

    hipMemcpy(out, d_C, sizeC, hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

// Cublas for matrix multiplication with A and transpose(B)
extern "C" void matMulCublas(float* a, int aRows, int aCols, float* b, int bRows, int bCols, float* out) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float *d_A, *d_B, *d_C;
    size_t sizeA = aRows * aCols * sizeof(float);
    size_t sizeB = bRows * bCols * sizeof(float);
    size_t sizeC = aRows * bRows * sizeof(float);

    hipMalloc((void**)&d_A, sizeA);
    hipMalloc((void**)&d_B, sizeB);
    hipMalloc((void**)&d_C, sizeC);

    hipMemcpy(d_A, a, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, b, sizeB, hipMemcpyHostToDevice);
    
    // Cublas Kernel
    float one = 1.0;
    float zero = 0.0;

    // We WTG C = A * B.T
    // Cublas stores in column order while C stores in row order
    // So Cublas interprets A and B as A.T and B.T
    // Therefore we input B.T * A -> interpreted as B * A.T = C.T
    // C.T in column major = C in row major, so we have what we want
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                bRows, aRows, aCols, // rows C, cols C, cols op(A)
                &one,
                d_B, bCols, // ld B
                d_A, aCols, // ld A
                &zero, d_C, bRows); // ld C

    hipMemcpy(out, d_C, sizeC, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);
}

// Take a slice out of a larger matrix and return a new matrix with the given shape
extern "C" Matrix sliceCublas(Matrix a, int b, int rows, int cols) {
    // change to devicetodevice later TODO
    Matrix out = {a.dat + b * rows, rows, cols};
    return out;
}

// From Lab 2
__global__
void transposeKernel(const float *input, float *output, int rows, int cols) {
    const int TILE_DIM = 64;
    __shared__ float tile[TILE_DIM][TILE_DIM + 1];  // +1 for padding to avoid bank conflicts

    // Global index calculations for reading input
    int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
    int yIndex = blockIdx.y * TILE_DIM + 4 * threadIdx.y;  // Each thread reads 4 elements along y

    // Local index within shared memory
    int localX = threadIdx.x;
    int localY = threadIdx.y;

    // Read input matrix in a coalesced manner and store into shared memory
    if (xIndex < cols) {
        if (yIndex + 0 < rows) tile[localY * 4 + 0][localX] = input[xIndex + (yIndex + 0) * cols];
        if (yIndex + 1 < rows) tile[localY * 4 + 1][localX] = input[xIndex + (yIndex + 1) * cols];
        if (yIndex + 2 < rows) tile[localY * 4 + 2][localX] = input[xIndex + (yIndex + 2) * cols];
        if (yIndex + 3 < rows) tile[localY * 4 + 3][localX] = input[xIndex + (yIndex + 3) * cols];
    }

    __syncthreads();  // Synchronize to ensure all writes to shared memory are complete

    // Transpose within shared memory
    xIndex = blockIdx.y * TILE_DIM + threadIdx.x;
    yIndex = blockIdx.x * TILE_DIM + 4 * threadIdx.y;

    // Write output in a coalesced manner
    if (xIndex < rows) {
        if (yIndex + 0 < cols) output[(yIndex + 0) * rows + xIndex] = tile[localX][localY * 4 + 0];
        if (yIndex + 1 < cols) output[(yIndex + 1) * rows + xIndex] = tile[localX][localY * 4 + 1];
        if (yIndex + 2 < cols) output[(yIndex + 2) * rows + xIndex] = tile[localX][localY * 4 + 2];
        if (yIndex + 3 < cols) output[(yIndex + 3) * rows + xIndex] = tile[localX][localY * 4 + 3];
    }
}

extern "C" void transposeCUDA(Matrix a, Matrix out)
{
    float *d_input, *d_output;
    size_t size = a.rows * a.cols * sizeof(float);

    hipMalloc(&d_input, size);
    hipMalloc(&d_output, size);

    hipMemcpy(d_input, a.dat, size, hipMemcpyHostToDevice);

    const int TILE_DIM = 64;

    dim3 dimBlock(TILE_DIM, TILE_DIM / 4); // 64x16
    dim3 dimGrid((a.cols + TILE_DIM - 1) / TILE_DIM, (a.rows + TILE_DIM - 1) / TILE_DIM);

    transposeKernel<<<dimGrid, dimBlock>>>(d_input, d_output, a.rows, a.cols);

    hipMemcpy(out.dat, d_output, size, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}

//  Matrix fn(Matrix a, float k)
#define UNARY(fn, opr)                                                 \
    __global__ void fn##Kernel(float* a, int aRows, int aCols, float* out, float k) { \
        int row = blockIdx.y * blockDim.y + threadIdx.y;               \
        int col = blockIdx.x * blockDim.x + threadIdx.x;               \
        if (row < aRows && col < aCols) {                              \
            int i = row * aCols + col;                                 \
            float b = a[i];                                            \
            out[i] = opr;                                              \
        }                                                              \
    }                                                                  \
    extern "C" Matrix fn##CUDA(Matrix m, float k) {                    \
        float* a = m.dat;                                              \
        int aRows = m.rows;                                            \
        int aCols = m.cols;                                            \
        float *d_a;                                                    \
        size_t sizeA = aRows * aCols * sizeof(float);                  \
        hipMalloc((void**)&d_a, sizeA);                               \
        hipMemcpy(d_a, a, sizeA, hipMemcpyHostToDevice);             \
        dim3 blockSize(16, 16);                                        \
        dim3 gridSize((aCols + blockSize.x - 1) / blockSize.x,         \
                      (aRows + blockSize.y - 1) / blockSize.y);        \
        fn##Kernel<<<gridSize, blockSize>>>(d_a, aRows, aCols, d_a, k);\
        hipMemcpy(m.dat, d_a, sizeA, hipMemcpyDeviceToHost);         \
        hipFree(d_a);                                                 \
        return m;                                                      \
    }

UNARY(divide_const, b / k)                      // divide by a constant
UNARY(add_const, b + k)                         // add a constant
UNARY(mat_isqrt, 1. / sqrt(b))                  // square root each entry
UNARY(mat_exp, exp(b))                          // exponetiate each entry
UNARY(broadcast, a[(i / aCols) * aCols])  // copy the first column to every column

// Tril is the first of two special functions.
//   a   b   c        exp(a/8) exp(b/8) exp(c/8)
//   d   e   f   ->      0     exp(e/8) exp(f/8)
//   g   h   i           0        0        0
// it's use will be described later
UNARY(tril, (i / k < i % (int)k) ? 0 : exp(b / 8))

// GELU is the activation function used for transformers
UNARY(GELU, b / 2 * (1 + tanh(.7978845 * (b + .044715 * b * b * b))))