#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblas.h>
#include "cuda_utils.h"

// CUDA kernel for matrix multiplication with A and transpose(B)
__global__ void matMulCudaKernel(float* A, float* B, float* C, int aRows, int aCols, int bRows) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < aRows && col < bRows) {
        float value = 0;
        for (int k = 0; k < aCols; ++k) {
            // B is accessed in transposed manner
            value += A[row * aCols + k] * B[col * aCols + k];
        }
        C[row * bRows + col] = value;
    }
}

extern "C" void matMulCUDA(float* a, int aRows, int aCols, float* b, int bRows, int bCols, float* out) {
    float *d_A, *d_B, *d_C;
    size_t sizeA = aRows * aCols * sizeof(float);
    size_t sizeB = bRows * bCols * sizeof(float);
    size_t sizeC = aRows * bRows * sizeof(float);

    hipMalloc((void**)&d_A, sizeA);
    hipMalloc((void**)&d_B, sizeB);
    hipMalloc((void**)&d_C, sizeC);

    hipMemcpy(d_A, a, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, b, sizeB, hipMemcpyHostToDevice);

    // Cuda Kernel
    dim3 dimBlock(16, 16);
    dim3 dimGrid((bRows + dimBlock.x - 1) / dimBlock.x, (aRows + dimBlock.y - 1) / dimBlock.y);
    matMulCudaKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, aRows, aCols, bRows);

    hipMemcpy(out, d_C, sizeC, hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

// Cublas for matrix multiplication with A and transpose(B)
extern "C" void matMulCublas(float* a, int aRows, int aCols, float* b, int bRows, int bCols, float* out) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float *d_A, *d_B, *d_C;
    size_t sizeA = aRows * aCols * sizeof(float);
    size_t sizeB = bRows * bCols * sizeof(float);
    size_t sizeC = aRows * bRows * sizeof(float);

    hipMalloc((void**)&d_A, sizeA);
    hipMalloc((void**)&d_B, sizeB);
    hipMalloc((void**)&d_C, sizeC);

    hipMemcpy(d_A, a, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, b, sizeB, hipMemcpyHostToDevice);
    
    // Cublas Kernel
    float one = 1.0;
    float zero = 0.0;

    // We WTG C = A * B.T
    // Cublas stores in column order while C stores in row order
    // So Cublas interprets A and B as A.T and B.T
    // Therefore we input B.T * A -> interpreted as B * A.T = C.T
    // C.T in column major = C in row major, so we have what we want
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                bRows, aRows, aCols, // rows C, cols C, cols op(A)
                &one,
                d_B, bCols, // ld B
                d_A, aCols, // ld A
                &zero, d_C, bRows); // ld C

    hipMemcpy(out, d_C, sizeC, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);
}

// Take a slice out of a larger matrix and return a new matrix with the given shape
extern "C" Matrix sliceCublas(Matrix a, int b, int rows, int cols) {
    // change to devicetodevice later TODO
    Matrix out = {a.dat + b * rows, rows, cols};
    return out;
}