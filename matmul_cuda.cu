#include <hip/hip_runtime.h>
#include <iostream>

// CUDA kernel for matrix multiplication with A and transpose(B)
__global__ void matMulKernel(float* A, float* B, float* C, int aRows, int aCols, int bRows) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < aRows && col < bRows) { // Notice the change from bCols to bRows
        float value = 0;
        for (int k = 0; k < aCols; ++k) {
            // B is accessed in transposed manner
            value += A[row * aCols + k] * B[col * aCols + k]; // Accessing B[col][k] instead of B[k][col]
        }
        C[row * bRows + col] = value; // bCols changed to bRows
    }
}

extern "C" void matMulCUDA(float* a, int aRows, int aCols, float* b, int bRows, int bCols, float* out) {
    if (aCols != bCols) { // Adjust the condition to check aCols against bCols, since B is transposed
        std::cerr << "Matrix dimensions mismatch: aCols must equal bCols for transpose multiplication." << std::endl;
        return;
    }

    float *d_A, *d_B, *d_C;
    size_t sizeA = aRows * aCols * sizeof(float);
    size_t sizeB = bRows * bCols * sizeof(float);
    size_t sizeC = aRows * bRows * sizeof(float); // bCols changed to bRows

    hipError_t status;
    status = hipMalloc((void**)&d_A, sizeA);
    if (status != hipSuccess) { std::cerr << "CUDA malloc failed for d_A" << std::endl; return; }
    status = hipMalloc((void**)&d_B, sizeB);
    if (status != hipSuccess) { std::cerr << "CUDA malloc failed for d_B" << std::endl; hipFree(d_A); return; }
    status = hipMalloc((void**)&d_C, sizeC);
    if (status != hipSuccess) { std::cerr << "CUDA malloc failed for d_C" << std::endl; hipFree(d_A); hipFree(d_B); return; }

    hipMemcpy(d_A, a, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, b, sizeB, hipMemcpyHostToDevice);

    dim3 dimBlock(16, 16);
    dim3 dimGrid((bRows + dimBlock.x - 1) / dimBlock.x, (aRows + dimBlock.y - 1) / dimBlock.y);

    matMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, aRows, aCols, bRows); // Changed bCols to bRows
    status = hipGetLastError();
    if (status != hipSuccess) { std::cerr << "Kernel launch failed: " << hipGetErrorString(status) << std::endl; hipFree(d_A); hipFree(d_B); hipFree(d_C); return; }

    hipMemcpy(out, d_C, sizeC, hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}