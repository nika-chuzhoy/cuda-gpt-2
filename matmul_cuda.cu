#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "matmul_cuda.h"

// CUDA kernel for matrix multiplication
__global__ void matMulKernel(float* A, float* B, float* C, int aRows, int aCols, int bCols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < aRows && col < bCols) {
        float value = 0;
        for (int k = 0; k < aCols; ++k) {
            value += A[row * aCols + k] * B[k * bCols + col];
        }
        C[row * bCols + col] = value;
    }
}

extern "C" void matMulCUDA(float* a, int aRows, int aCols, float* b, int bRows, int bCols, float* out) {
    float *d_A, *d_B, *d_C;
    size_t sizeA = aRows * aCols * sizeof(float);
    size_t sizeB = bRows * bCols * sizeof(float);
    size_t sizeC = aRows * bCols * sizeof(float);

    hipMalloc((void**)&d_A, sizeA);
    hipMalloc((void**)&d_B, sizeB);
    hipMalloc((void**)&d_C, sizeC);

    hipMemcpy(d_A, a, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, b, sizeB, hipMemcpyHostToDevice);

    dim3 dimBlock(16, 16);
    dim3 dimGrid((bCols + dimBlock.x - 1) / dimBlock.x, (aRows + dimBlock.y - 1) / dimBlock.y);

    matMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, aRows, aCols, bCols);

    hipMemcpy(out, d_C, sizeC, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
